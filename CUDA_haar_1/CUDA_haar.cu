
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
//#include "opencv2/objdetect.hpp"
#include "opencv2/cudaobjdetect.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/imgproc.hpp"
#include <iostream>


using namespace std;
using namespace cv;

void detect_Display(Mat frame);

//cuda::CascadeClassifier model_cascade;
Ptr<cuda::CascadeClassifier> cascade_gpu;// = cuda::CascadeClassifier::create("E:/University/10sem/nirs/haar_3_4_6/x64/Debug/cascade.xml");

int main(int argc, const char** argv) {

	CommandLineParser parser(argc, argv,
		"{help h||}"
		"{model_cascade|E:/University/10sem/nirs/haar_3_4_6/x64/Debug/cascade.xml|Path to face cascade.}"
		"{camera|0|Camera device number.}");
	parser.about("\nThis program demonstrates using the cv::CascadeClassifier class to detect objects in a video stream.\n"
		"You can use Haar or LBP features.\n\n");
	parser.printMessage();

	String model_cascade_name = parser.get<String>("model_cascade");

	//-- 1. Load the cascades
	if (!cascade_gpu->create(model_cascade_name)) {
		cout << "--(!)Error loading face cascade\n";
		return -1;
	};

	int camera_device = parser.get<int>("camera");
	VideoCapture capture(camera_device);

	//-- 2. Read the video stream
	//capture.open(camera_device);

	if (!capture.isOpened()) {
		cout << "--(!)Error opening video capture\n";
		return -1;
	}
	Mat frame;

	while (1) {

		capture.read(frame);

		if (frame.empty()) {
			cout << "--(!) No captured frame -- Break!\n";
			break;
		}
		imshow("Captured", frame);

		//-- 3. Apply the classifier to the frame
		detect_Display(frame);

		// Press  ESC on keyboard to exit
		if (waitKey(25) == 27)
			break;
	}
	// When everything done, release the video capture object
	capture.release();

	// Closes all the frames
	destroyAllWindows();

	return 0;


	hipError_t cudaStatus;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

